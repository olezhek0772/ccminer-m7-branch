#if 1

#include <hip/hip_runtime.h>



#include <stdio.h>
#include <memory.h>

// Folgende Definitionen sp�ter durch header ersetzen
//////typedef unsigned int uint32_t;
//typedef unsigned int uint32_t;

// Endian Drehung f�r 32 Bit Typen
/*
static __device__ uint32_t cuda_swab32(uint32_t x)
{
    return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
          | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}
*/
static __device__ uint32_t cuda_swab32(uint32_t x)
{
	return __byte_perm(x, 0, 0x0123);
}
// Endian Drehung f�r 64 Bit Typen
static __device__ unsigned long long cuda_swab64(unsigned long long x) {
    uint32_t h = (x >> 32);
    uint32_t l = (x & 0xFFFFFFFFULL);
    return (((unsigned long long)cuda_swab32(l)) << 32) | ((unsigned long long)cuda_swab32(h));
}

// das Hi Word aus einem 64 Bit Typen extrahieren
static __device__ uint32_t HIWORD(const unsigned long long &x) {
#if __CUDA_ARCH__ >= 130
	return (uint32_t)__double2hiint(__longlong_as_double(x));
#else
	return (uint32_t)(x >> 32);
#endif
}

// das Hi Word in einem 64 Bit Typen ersetzen
static __device__ unsigned long long REPLACE_HIWORD(const unsigned long long &x, const uint32_t &y) {
	return (x & 0xFFFFFFFFULL) | (((unsigned long long)y) << 32ULL);
}

// das Lo Word aus einem 64 Bit Typen extrahieren
static __device__ uint32_t LOWORD(const unsigned long long &x) {
#if __CUDA_ARCH__ >= 130
	return (uint32_t)__double2loint(__longlong_as_double(x));
#else
	return (uint32_t)(x & 0xFFFFFFFFULL);
#endif
}

static __device__ unsigned long long MAKE_ULONGLONG(uint32_t LO, uint32_t HI)
{
#if __CUDA_ARCH__ >= 130
    return __double_as_longlong(__hiloint2double(HI, LO));
#else
	return (unsigned long long)LO | (((unsigned long long)HI) << 32ULL);
#endif
}

// das Lo Word in einem 64 Bit Typen ersetzen
static __device__ unsigned long long REPLACE_LOWORD(const unsigned long long &x, const uint32_t &y) {
	return (x & 0xFFFFFFFF00000000ULL) | ((unsigned long long)y);
}

// der Versuch, einen Wrapper f�r einen aus 32 Bit Registern zusammengesetzten uin64_t Typen zu entferfen...
#if 1
//typedef unsigned long long uint64_t;
#else
typedef class uint64
{
public:
	__device__ uint64()
	{
	}
	__device__ uint64(unsigned long long init)
	{
		val = make_uint2( LOWORD(init), HIWORD(init) );
	}
	__device__ uint64(uint32_t lo, uint32_t hi)
	{
		val = make_uint2( lo, hi );
	}
	__device__ const uint64 operator^(uint64 const& rhs) const
	{
		return uint64(val.x ^ rhs.val.x, val.y ^ rhs.val.y);
	}
	__device__ const uint64 operator|(uint64 const& rhs) const
	{
		return uint64(val.x | rhs.val.x, val.y | rhs.val.y);
	}
	__device__ const uint64 operator+(unsigned long long const& rhs) const
	{
		return *this+uint64(rhs);
	}
	__device__ const uint64 operator+(uint64 const& rhs) const
	{
		uint64 res;
		asm ("add.cc.u32      %0, %2, %4;\n\t"
			 "addc.cc.u32     %1, %3, %5;\n\t"
			 : "=r"(res.val.x), "=r"(res.val.y)
			 : "r"(    val.x), "r"(    val.y),
			   "r"(rhs.val.x), "r"(rhs.val.y));
		return res;
	}
	__device__ const uint64 operator-(uint64 const& rhs) const
	{
		uint64 res;
		asm ("sub.cc.u32      %0, %2, %4;\n\t"
			 "subc.cc.u32     %1, %3, %5;\n\t"
			 : "=r"(res.val.x), "=r"(res.val.y)
			 : "r"(    val.x), "r"(    val.y),
			   "r"(rhs.val.x), "r"(rhs.val.y));
		return res;
	}
	__device__ const uint64 operator<<(int n) const
	{
		return uint64(unsigned long long(*this)<<n);
	}
	__device__ const uint64 operator>>(int n) const
	{
		return uint64(unsigned long long(*this)>>n);
	}
	__device__ operator unsigned long long() const
	{
		return MAKE_ULONGLONG(val.x, val.y);
	}
	uint2 val;
} uint64_t;
#endif

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// die Message it Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

#define SPH_C64(x)    ((uint64_t)(x ## ULL))

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// diese 64 Bit Rotates werden unter Compute 3.5 (und besser) mit dem Funnel Shifter beschleunigt
#if __CUDA_ARCH__ >= 350
__forceinline__ __device__ uint64_t ROTL64(const uint64_t value, const int offset) {
    uint2 result;
    if(offset >= 32) {
        asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
        asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
    } else {
        asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
        asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
    }
    return  __double_as_longlong(__hiloint2double(result.y, result.x));
}
#else
#define ROTL64(x, n)        (((x) << (n)) | ((x) >> (64 - (n))))
#endif
#define SHL(x, n)            ((x) << (n))
#define SHR(x, n)            ((x) >> (n))

#define CONST_EXP2    q[i+0] + ROTL64(q[i+1], 5)  + q[i+2] + ROTL64(q[i+3], 11) + \
                    q[i+4] + ROTL64(q[i+5], 27) + q[i+6] + ROTL64(q[i+7], 32) + \
                    q[i+8] + ROTL64(q[i+9], 37) + q[i+10] + ROTL64(q[i+11], 43) + \
                    q[i+12] + ROTL64(q[i+13], 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])

__device__ void Compression512(uint64_t *msg, uint64_t *hash)
{
    // Compression ref. implementation
    uint64_t tmp;
    uint64_t q[32];

    tmp = (msg[ 5] ^ hash[ 5]) - (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]) + (msg[14] ^ hash[14]);
    q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[1];
    tmp = (msg[ 6] ^ hash[ 6]) - (msg[ 8] ^ hash[ 8]) + (msg[11] ^ hash[11]) + (msg[14] ^ hash[14]) - (msg[15] ^ hash[15]);
    q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[2];
    tmp = (msg[ 0] ^ hash[ 0]) + (msg[ 7] ^ hash[ 7]) + (msg[ 9] ^ hash[ 9]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[3];
    tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 1] ^ hash[ 1]) + (msg[ 8] ^ hash[ 8]) - (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]);
    q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[4];
    tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 2] ^ hash[ 2]) + (msg[ 9] ^ hash[ 9]) - (msg[11] ^ hash[11]) - (msg[14] ^ hash[14]);
    q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
    tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 2] ^ hash[ 2]) + (msg[10] ^ hash[10]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[6];
    tmp = (msg[ 4] ^ hash[ 4]) - (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) - (msg[11] ^ hash[11]) + (msg[13] ^ hash[13]);
    q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[7];
    tmp = (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 5] ^ hash[ 5]) - (msg[12] ^ hash[12]) - (msg[14] ^ hash[14]);
    q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[8];
    tmp = (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) - (msg[ 6] ^ hash[ 6]) + (msg[13] ^ hash[13]) - (msg[15] ^ hash[15]);
    q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[9];
    tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) + (msg[ 6] ^ hash[ 6]) - (msg[ 7] ^ hash[ 7]) + (msg[14] ^ hash[14]);
    q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
    tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 7] ^ hash[ 7]) + (msg[15] ^ hash[15]);
    q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[11];
    tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 0] ^ hash[ 0]) - (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) + (msg[ 9] ^ hash[ 9]);
    q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[12];
    tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 3] ^ hash[ 3]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[10] ^ hash[10]);
    q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[13];
    tmp = (msg[ 2] ^ hash[ 2]) + (msg[ 4] ^ hash[ 4]) + (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[11] ^ hash[11]);
    q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[14];
    tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 5] ^ hash[ 5]) + (msg[ 8] ^ hash[ 8]) - (msg[11] ^ hash[11]) - (msg[12] ^ hash[12]);
    q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
    tmp = (msg[12] ^ hash[12]) - (msg[ 4] ^ hash[ 4]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[13] ^ hash[13]);
    q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[0];

    // Expand 1
#pragma unroll 2
    for(int i=0;i<2;i++)
    {
        q[i+16] =
        (SHR(q[i], 1) ^ SHL(q[i], 2) ^ ROTL64(q[i], 13) ^ ROTL64(q[i], 43)) +
        (SHR(q[i+1], 2) ^ SHL(q[i+1], 1) ^ ROTL64(q[i+1], 19) ^ ROTL64(q[i+1], 53)) +
        (SHR(q[i+2], 2) ^ SHL(q[i+2], 2) ^ ROTL64(q[i+2], 28) ^ ROTL64(q[i+2], 59)) +
        (SHR(q[i+3], 1) ^ SHL(q[i+3], 3) ^ ROTL64(q[i+3],  4) ^ ROTL64(q[i+3], 37)) +
        (SHR(q[i+4], 1) ^ SHL(q[i+4], 2) ^ ROTL64(q[i+4], 13) ^ ROTL64(q[i+4], 43)) +
        (SHR(q[i+5], 2) ^ SHL(q[i+5], 1) ^ ROTL64(q[i+5], 19) ^ ROTL64(q[i+5], 53)) +
        (SHR(q[i+6], 2) ^ SHL(q[i+6], 2) ^ ROTL64(q[i+6], 28) ^ ROTL64(q[i+6], 59)) +
        (SHR(q[i+7], 1) ^ SHL(q[i+7], 3) ^ ROTL64(q[i+7],  4) ^ ROTL64(q[i+7], 37)) +
        (SHR(q[i+8], 1) ^ SHL(q[i+8], 2) ^ ROTL64(q[i+8], 13) ^ ROTL64(q[i+8], 43)) +
        (SHR(q[i+9], 2) ^ SHL(q[i+9], 1) ^ ROTL64(q[i+9], 19) ^ ROTL64(q[i+9], 53)) +
        (SHR(q[i+10], 2) ^ SHL(q[i+10], 2) ^ ROTL64(q[i+10], 28) ^ ROTL64(q[i+10], 59)) +
        (SHR(q[i+11], 1) ^ SHL(q[i+11], 3) ^ ROTL64(q[i+11],  4) ^ ROTL64(q[i+11], 37)) +
        (SHR(q[i+12], 1) ^ SHL(q[i+12], 2) ^ ROTL64(q[i+12], 13) ^ ROTL64(q[i+12], 43)) +
        (SHR(q[i+13], 2) ^ SHL(q[i+13], 1) ^ ROTL64(q[i+13], 19) ^ ROTL64(q[i+13], 53)) +
        (SHR(q[i+14], 2) ^ SHL(q[i+14], 2) ^ ROTL64(q[i+14], 28) ^ ROTL64(q[i+14], 59)) +
        (SHR(q[i+15], 1) ^ SHL(q[i+15], 3) ^ ROTL64(q[i+15],  4) ^ ROTL64(q[i+15], 37)) +
        ((    ((i+16)*(0x0555555555555555ull)) + ROTL64(msg[i], i+1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i+10], i+11) ) ^ hash[i+7]);
    }

#pragma unroll 4
    for(int i=2;i<6;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    ((i+16)*(0x0555555555555555ull)) + ROTL64(msg[i], i+1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i+10], i+11) ) ^ hash[i+7]);
    }
#pragma unroll 3
    for(int i=6;i<9;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    ((i+16)*(0x0555555555555555ull)) + ROTL64(msg[i], i+1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i-6], (i-6)+1) ) ^ hash[i+7]);
    }
#pragma unroll 4
    for(int i=9;i<13;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    ((i+16)*(0x0555555555555555ull)) + ROTL64(msg[i], i+1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i-6], (i-6)+1) ) ^ hash[i-9]);
    }
#pragma unroll 3
    for(int i=13;i<16;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    ((i+16)*(0x0555555555555555ull)) + ROTL64(msg[i], i+1) +
            ROTL64(msg[i-13], (i-13)+1) - ROTL64(msg[i-6], (i-6)+1) ) ^ hash[i-9]);
    }

    uint64_t XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
    uint64_t XH64 = XL64^q[24]^q[25]^q[26]^q[27]^q[28]^q[29]^q[30]^q[31];

    hash[0] =                       (SHL(XH64, 5) ^ SHR(q[16],5) ^ msg[ 0]) + (    XL64    ^ q[24] ^ q[ 0]);
    hash[1] =                       (SHR(XH64, 7) ^ SHL(q[17],8) ^ msg[ 1]) + (    XL64    ^ q[25] ^ q[ 1]);
    hash[2] =                       (SHR(XH64, 5) ^ SHL(q[18],5) ^ msg[ 2]) + (    XL64    ^ q[26] ^ q[ 2]);
    hash[3] =                       (SHR(XH64, 1) ^ SHL(q[19],5) ^ msg[ 3]) + (    XL64    ^ q[27] ^ q[ 3]);
    hash[4] =                       (SHR(XH64, 3) ^     q[20]    ^ msg[ 4]) + (    XL64    ^ q[28] ^ q[ 4]);
    hash[5] =                       (SHL(XH64, 6) ^ SHR(q[21],6) ^ msg[ 5]) + (    XL64    ^ q[29] ^ q[ 5]);
    hash[6] =                       (SHR(XH64, 4) ^ SHL(q[22],6) ^ msg[ 6]) + (    XL64    ^ q[30] ^ q[ 6]);
    hash[7] =                       (SHR(XH64,11) ^ SHL(q[23],2) ^ msg[ 7]) + (    XL64    ^ q[31] ^ q[ 7]);

    hash[ 8] = ROTL64(hash[4], 9) + (    XH64     ^     q[24]    ^ msg[ 8]) + (SHL(XL64,8) ^ q[23] ^ q[ 8]);
    hash[ 9] = ROTL64(hash[5],10) + (    XH64     ^     q[25]    ^ msg[ 9]) + (SHR(XL64,6) ^ q[16] ^ q[ 9]);
    hash[10] = ROTL64(hash[6],11) + (    XH64     ^     q[26]    ^ msg[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
    hash[11] = ROTL64(hash[7],12) + (    XH64     ^     q[27]    ^ msg[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
    hash[12] = ROTL64(hash[0],13) + (    XH64     ^     q[28]    ^ msg[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
    hash[13] = ROTL64(hash[1],14) + (    XH64     ^     q[29]    ^ msg[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
    hash[14] = ROTL64(hash[2],15) + (    XH64     ^     q[30]    ^ msg[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
    hash[15] = ROTL64(hash[3],16) + (    XH64     ^     q[31]    ^ msg[15]) + (SHR(XL64,2) ^ q[22] ^ q[15]);
}
static __constant__ uint64_t d_constMem[16];
static uint64_t h_constMem[16] = {
	SPH_C64(0x8081828384858687),
    SPH_C64(0x88898A8B8C8D8E8F),
    SPH_C64(0x9091929394959697),
    SPH_C64(0x98999A9B9C9D9E9F),
    SPH_C64(0xA0A1A2A3A4A5A6A7),
    SPH_C64(0xA8A9AAABACADAEAF),
    SPH_C64(0xB0B1B2B3B4B5B6B7),
    SPH_C64(0xB8B9BABBBCBDBEBF),
    SPH_C64(0xC0C1C2C3C4C5C6C7),
    SPH_C64(0xC8C9CACBCCCDCECF),
    SPH_C64(0xD0D1D2D3D4D5D6D7),
    SPH_C64(0xD8D9DADBDCDDDEDF),
    SPH_C64(0xE0E1E2E3E4E5E6E7),
    SPH_C64(0xE8E9EAEBECEDEEEF),
    SPH_C64(0xF0F1F2F3F4F5F6F7),
    SPH_C64(0xF8F9FAFBFCFDFEFF)
};

__global__ void quark_bmw512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint64_t *inpHash = &g_hash[8 * hashPosition];

        // Init
        uint64_t h[16];
		/*
        h[ 0] = SPH_C64(0x8081828384858687);
        h[ 1] = SPH_C64(0x88898A8B8C8D8E8F);
        h[ 2] = SPH_C64(0x9091929394959697);
        h[ 3] = SPH_C64(0x98999A9B9C9D9E9F);
        h[ 4] = SPH_C64(0xA0A1A2A3A4A5A6A7);
        h[ 5] = SPH_C64(0xA8A9AAABACADAEAF);
        h[ 6] = SPH_C64(0xB0B1B2B3B4B5B6B7);
        h[ 7] = SPH_C64(0xB8B9BABBBCBDBEBF);
        h[ 8] = SPH_C64(0xC0C1C2C3C4C5C6C7);
        h[ 9] = SPH_C64(0xC8C9CACBCCCDCECF);
        h[10] = SPH_C64(0xD0D1D2D3D4D5D6D7);
        h[11] = SPH_C64(0xD8D9DADBDCDDDEDF);
        h[12] = SPH_C64(0xE0E1E2E3E4E5E6E7);
        h[13] = SPH_C64(0xE8E9EAEBECEDEEEF);
        h[14] = SPH_C64(0xF0F1F2F3F4F5F6F7);
        h[15] = SPH_C64(0xF8F9FAFBFCFDFEFF);
		*/
#pragma unroll 16
		for(int i=0;i<16;i++)
			h[i] = d_constMem[i];
        // Nachricht kopieren (Achtung, die Nachricht hat 64 Byte,
        // BMW arbeitet mit 128 Byte!!!
        uint64_t message[16];
#pragma unroll 8
        for(int i=0;i<8;i++)
            message[i] = inpHash[i];
#pragma unroll 6
        for(int i=9;i<15;i++)
            message[i] = 0;

        // Padding einf�gen (Byteorder?!?)
        message[8] = SPH_C64(0x80);
        // L�nge (in Bits, d.h. 64 Byte * 8 = 512 Bits
        message[15] = SPH_C64(512);

        // Compression 1
        Compression512(message, h);

        // Final
#pragma unroll 16
        for(int i=0;i<16;i++)
            message[i] = 0xaaaaaaaaaaaaaaa0ull + (uint64_t)i;

        Compression512(h, message);

        // fertig
        uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
        for(int i=0;i<8;i++)
            outpHash[i] = message[i+8];
    }
}

__global__ void quark_bmw512_gpu_hash_80(int threads, uint32_t startNounce, uint64_t *g_hash)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = startNounce + thread;

        // Init
        uint64_t h[16];
#pragma unroll 16
		for(int i=0;i<16;i++)
			h[i] = d_constMem[i];

        // Nachricht kopieren (Achtung, die Nachricht hat 64 Byte,
        // BMW arbeitet mit 128 Byte!!!
        uint64_t message[16];
#pragma unroll 16
        for(int i=0;i<16;i++)
            message[i] = c_PaddedMessage80[i];

        // die Nounce durch die thread-spezifische ersetzen
        message[9] = REPLACE_HIWORD(message[9], cuda_swab32(nounce));

        // Compression 1
        Compression512(message, h);

        // Final
#pragma unroll 16
        for(int i=0;i<16;i++)
            message[i] = 0xaaaaaaaaaaaaaaa0ull + (uint64_t)i;

        Compression512(h, message);

        // fertig
        uint64_t *outpHash = &g_hash[8 * thread];

#pragma unroll 8
        for(int i=0;i<8;i++)
            outpHash[i] = message[i+8];
    }
}

// Setup-Funktionen
__host__ void quark_bmw512_cpu_init(int thr_id, int threads)
{
    // nix zu tun ;-)
	// jetzt schon :D
	hipMemcpyToSymbol( HIP_SYMBOL(d_constMem),
                        h_constMem,
                        sizeof(h_constMem),
                        0, hipMemcpyHostToDevice);
}

// Bmw512 f�r 80 Byte grosse Eingangsdaten
__host__ void quark_bmw512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	uint64_t *message = (uint64_t*)PaddedMessage;
	// Padding einf�gen (Byteorder?!?)
	message[10] = SPH_C64(0x80);
	// L�nge (in Bits, d.h. 80 Byte * 8 = 640 Bits
	message[15] = SPH_C64(640);

	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol( HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__ void quark_bmw512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    quark_bmw512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void quark_bmw512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order)
{
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    quark_bmw512_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash);
    MyStreamSynchronize(NULL, order, thr_id);
}

#endif
