#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

// Folgende Definitionen sp�ter durch header ersetzen
//////typedef unsigned int uint32_t;
//typedef unsigned int uint32_t;
//typedef unsigned long long uint64_t;

// das Hash Target gegen das wir testen sollen
__constant__ uint32_t pTarget[8];

uint32_t *d_resNounce[8];
uint32_t *h_resNounce[8];

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__global__ void quark_check_gpu_hash_64(int threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Z�hler
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint32_t *inpHash = &g_hash[16 * hashPosition];

		uint32_t hash[8];
#pragma unroll 8
		for (int i=0; i < 8; i++)
			hash[i] = inpHash[i];

		// kopiere Ergebnis
		int i, position = -1;
		bool rc = true;

#pragma unroll 8
		for (i = 7; i >= 0; i--) {
			if (hash[i] > pTarget[i]) {
				if(position < i) {
					position = i;
					rc = false;
				}
	 		}
	 		if (hash[i] < pTarget[i]) {
				if(position < i) {
					position = i;
					rc = true;
				}
	 		}
		}

		if(rc == true)
			if(resNounce[0] > nounce)
				resNounce[0] = nounce;
	}
}

// Setup-Funktionen
__host__ void quark_check_cpu_init(int thr_id, int threads)
{
    hipHostMalloc(&h_resNounce[thr_id], 1*sizeof(uint32_t));
    hipMalloc(&d_resNounce[thr_id], 1*sizeof(uint32_t));
}

// Target Difficulty setzen
__host__ void quark_check_cpu_setTarget(const void *ptarget)
{
	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol(HIP_SYMBOL( pTarget), ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

__host__ uint32_t quark_check_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order)
{
	uint32_t result = 0xffffffff;
	hipMemset(d_resNounce[thr_id], 0xff, sizeof(uint32_t));

	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	quark_check_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_nonceVector, d_inputHash, d_resNounce[thr_id]);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);

	// Ergebnis zum Host kopieren (in page locked memory, damits schneller geht)
	hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	// hipMemcpy() ist asynchron!
	hipDeviceSynchronize();
	result = *h_resNounce[thr_id];

	return result;
}
