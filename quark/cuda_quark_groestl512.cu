#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

// aus cpu-miner.c
extern int device_map[8];

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// Folgende Definitionen sp�ter durch header ersetzen
//////typedef unsigned int uint32_t;
typedef unsigned short uint16_t;
//typedef unsigned int uint32_t;

// diese Struktur wird in der Init Funktion angefordert
static hipDeviceProp_t props[8];

// 64 Register Variante f�r Compute 3.0
#include "groestl_functions_quad.cu"
#include "bitslice_transformations_quad.cu"

__global__ void __launch_bounds__(256, 4)
 quark_groestl512_gpu_hash_64_quad(int threads, uint32_t startNounce, uint32_t *g_hash, uint32_t *g_nonceVector)
{
    // durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
    if (thread < threads)
    {
        // GROESTL
        uint32_t message[8];
        uint32_t state[8];

        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *inpHash = &g_hash[hashPosition<<4];

#pragma unroll 4
        for(int k=0;k<4;k++) message[k] = inpHash[(k<<2) + (threadIdx.x&0x03)];
#pragma unroll 4
        for(int k=4;k<8;k++) message[k] = 0;

        if ((threadIdx.x&0x03) == 0) message[4] = 0x80;
        if ((threadIdx.x&0x03) == 3) message[7] = 0x01000000;

        uint32_t msgBitsliced[8];
        to_bitslice_quad(message, msgBitsliced);

        groestl512_progressMessage_quad(state, msgBitsliced);

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t *outpHash = &g_hash[hashPosition<<4];
        uint32_t hash[16];
        from_bitslice_quad(state, hash);

        if ((threadIdx.x & 0x03) == 0)
        {
#pragma unroll 16
            for(int k=0;k<16;k++) outpHash[k] = hash[k];
        }
    }
}

__global__ void __launch_bounds__(256, 4)
 quark_doublegroestl512_gpu_hash_64_quad(int threads, uint32_t startNounce, uint32_t *g_hash, uint32_t *g_nonceVector)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x)>>2;
    if (thread < threads)
    {
        // GROESTL
        uint32_t message[8];
        uint32_t state[8];

        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *inpHash = &g_hash[hashPosition<<4];

#pragma unroll 4
        for(int k=0;k<4;k++) message[k] = inpHash[(k<<2)+(threadIdx.x&0x03)];
#pragma unroll 4
        for(int k=4;k<8;k++) message[k] = 0;

        if ((threadIdx.x&0x03) == 0) message[4] = 0x80;
        if ((threadIdx.x&0x03) == 3) message[7] = 0x01000000;

        uint32_t msgBitsliced[8];
        to_bitslice_quad(message, msgBitsliced);

        for (int round=0; round<2; round++)
        {
            groestl512_progressMessage_quad(state, msgBitsliced);

            if (round < 1)
            {
                // Verkettung zweier Runden inclusive Padding.
                msgBitsliced[ 0] = __byte_perm(state[ 0], 0x00800100, 0x4341 + (((threadIdx.x%4)==3)<<13));
                msgBitsliced[ 1] = __byte_perm(state[ 1], 0x00800100, 0x4341);
                msgBitsliced[ 2] = __byte_perm(state[ 2], 0x00800100, 0x4341);
                msgBitsliced[ 3] = __byte_perm(state[ 3], 0x00800100, 0x4341);
                msgBitsliced[ 4] = __byte_perm(state[ 4], 0x00800100, 0x4341);
                msgBitsliced[ 5] = __byte_perm(state[ 5], 0x00800100, 0x4341);
                msgBitsliced[ 6] = __byte_perm(state[ 6], 0x00800100, 0x4341);
                msgBitsliced[ 7] = __byte_perm(state[ 7], 0x00800100, 0x4341 + (((threadIdx.x%4)==0)<<4));
            }
        }

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t *outpHash = &g_hash[hashPosition<<4];
        uint32_t hash[16];
        from_bitslice_quad(state, hash);

        if ((threadIdx.x & 0x03) == 0)
        {
#pragma unroll 16
            for(int k=0;k<16;k++) outpHash[k] = hash[k];
        }
    }
}

// Setup-Funktionen
__host__ void quark_groestl512_cpu_init(int thr_id, int threads)
{
    hipGetDeviceProperties(&props[thr_id], device_map[thr_id]);
}

__host__ void quark_groestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    int threadsperblock = 256;

    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    const int factor = 4;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    quark_groestl512_gpu_hash_64_quad<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void quark_doublegroestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    int threadsperblock = 256;

    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    const int factor = 4;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    quark_doublegroestl512_gpu_hash_64_quad<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, order, thr_id);
}
