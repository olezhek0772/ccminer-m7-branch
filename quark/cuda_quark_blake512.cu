#include <hip/hip_runtime.h>



#include <stdio.h>
#include <memory.h>

#define USE_SHUFFLE 0

// Folgende Definitionen sp�ter durch header ersetzen
//////typedef unsigned int uint32_t;
//typedef unsigned int uint32_t;
//typedef unsigned long long uint64_t;

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// die Message it Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

// ---------------------------- BEGIN CUDA quark_blake512 functions ------------------------------------

__constant__ uint8_t c_sigma[16][16];

const uint8_t host_sigma[16][16] =
{
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
  {12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
  {13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
  { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
  {10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

// das Hi Word aus einem 64 Bit Typen extrahieren
static __device__ uint32_t HIWORD(const uint64_t &x) {
#if __CUDA_ARCH__ >= 130
	return (uint32_t)__double2hiint(__longlong_as_double(x));
#else
	return (uint32_t)(x >> 32);
#endif
}

// das Hi Word in einem 64 Bit Typen ersetzen
static __device__ uint64_t REPLACE_HIWORD(const uint64_t &x, const uint32_t &y) {
	return (x & 0xFFFFFFFFULL) | (((uint64_t)y) << 32ULL);
}

// das Lo Word aus einem 64 Bit Typen extrahieren
static __device__ uint32_t LOWORD(const uint64_t &x) {
#if __CUDA_ARCH__ >= 130
	return (uint32_t)__double2loint(__longlong_as_double(x));
#else
	return (uint32_t)(x & 0xFFFFFFFFULL);
#endif
}

// das Lo Word in einem 64 Bit Typen ersetzen
static __device__ uint64_t REPLACE_LOWORD(const uint64_t &x, const uint32_t &y) {
	return (x & 0xFFFFFFFF00000000ULL) | ((uint64_t)y);
}

__device__ __forceinline__ uint64_t SWAP64(uint64_t x)
{
	// Input:	77665544 33221100
	// Output:	00112233 44556677
	uint64_t temp[2];
	temp[0] = __byte_perm(HIWORD(x), 0, 0x0123);
	temp[1] = __byte_perm(LOWORD(x), 0, 0x0123);

	return temp[0] | (temp[1]<<32);
}

__constant__ uint64_t c_u512[16];

const uint64_t host_u512[16] =
{
  0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL, 
  0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
  0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL, 
  0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
  0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL, 
  0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
  0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL, 
  0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
};


// diese 64 Bit Rotates werden unter Compute 3.5 (und besser) mit dem Funnel Shifter beschleunigt
#if __CUDA_ARCH__ >= 350
__forceinline__ __device__ uint64_t ROTR(const uint64_t value, const int offset) {
    uint2 result;
    if(offset < 32) {
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
    } else {
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
        asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
    }
    return  __double_as_longlong(__hiloint2double(result.y, result.x));
}
#else
#define ROTR(x, n)        (((x) >> (n)) | ((x) << (64 - (n))))
#endif

#define G(a,b,c,d,e)          \
    v[a] += (m[sigma[i][e]] ^ u512[sigma[i][e+1]]) + v[b];\
    v[d] = ROTR( v[d] ^ v[a],32);        \
    v[c] += v[d];           \
    v[b] = ROTR( v[b] ^ v[c],25);        \
    v[a] += (m[sigma[i][e+1]] ^ u512[sigma[i][e]])+v[b];  \
    v[d] = ROTR( v[d] ^ v[a],16);        \
    v[c] += v[d];           \
    v[b] = ROTR( v[b] ^ v[c],11);


__device__ void quark_blake512_compress( uint64_t *h, const uint64_t *block, const uint8_t ((*sigma)[16]), const uint64_t *u512, const int bits )
{
    uint64_t v[16], m[16], i;

#pragma unroll 16
    for( i = 0; i < 16; ++i )
    {
        m[i] = SWAP64(block[i]);
    }

#pragma unroll 8
    for( i = 0; i < 8; ++i )  v[i] = h[i];

    v[ 8] = u512[0];
    v[ 9] = u512[1];
    v[10] = u512[2];
    v[11] = u512[3];
    v[12] = u512[4];
    v[13] = u512[5];
    v[14] = u512[6];
    v[15] = u512[7];

    v[12] ^= bits;
    v[13] ^= bits;

//#pragma unroll 16
    for( i = 0; i < 16; ++i )
    {
        /* column step */
        G( 0, 4, 8, 12, 0 );
        G( 1, 5, 9, 13, 2 );
        G( 2, 6, 10, 14, 4 );
        G( 3, 7, 11, 15, 6 );
        /* diagonal step */
        G( 0, 5, 10, 15, 8 );
        G( 1, 6, 11, 12, 10 );
        G( 2, 7, 8, 13, 12 );
        G( 3, 4, 9, 14, 14 );
    }

#pragma unroll 16
    for( i = 0; i < 16; ++i )  h[i % 8] ^= v[i];
}

// Endian Drehung f�r 32 Bit Typen

static __device__ uint32_t cuda_swab32(uint32_t x)
{
	return __byte_perm(x, 0, 0x0123);
}

/*
// Endian Drehung f�r 64 Bit Typen
static __device__ uint64_t cuda_swab64(uint64_t x) {
    uint32_t h = (x >> 32);
    uint32_t l = (x & 0xFFFFFFFFULL);
    return (((uint64_t)cuda_swab32(l)) << 32) | ((uint64_t)cuda_swab32(h));
}
*/

static __constant__ uint64_t d_constMem[8];
static const uint64_t h_constMem[8] = {
	0x6a09e667f3bcc908ULL,
	0xbb67ae8584caa73bULL,
	0x3c6ef372fe94f82bULL,
	0xa54ff53a5f1d36f1ULL,
	0x510e527fade682d1ULL,
	0x9b05688c2b3e6c1fULL,
	0x1f83d9abfb41bd6bULL,
	0x5be0cd19137e2179ULL };

// Hash-Padding
static __constant__ uint64_t d_constHashPadding[8];
static const uint64_t h_constHashPadding[8] = {
	0x0000000000000080ull,
	0,
	0,
	0,
	0,
	0x0100000000000000ull,
	0,
	0x0002000000000000ull };

__global__ __launch_bounds__(256, 2) void quark_blake512_gpu_hash_64(int threads, uint32_t startNounce, uint32_t *g_nonceVector, uint64_t *g_hash)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

#if USE_SHUFFLE
	const int warpID = threadIdx.x & 0x0F; // 16 warps
	const int warpBlockID = (thread + 15)>>4; // aufrunden auf volle Warp-Bl�cke
	const int maxHashPosition = thread<<3;
#endif

#if USE_SHUFFLE
	if (warpBlockID < ( (threads+15)>>4 ))
#else
	if (thread < threads)
#endif
	{
		// bestimme den aktuellen Z�hler
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		//uint64_t *inpHash = &g_hash[8 * hashPosition];
		uint64_t *inpHash = &g_hash[hashPosition<<3];

		// State vorbereiten
		uint64_t h[8];
		/*
		h[0] = 0x6a09e667f3bcc908ULL;
		h[1] = 0xbb67ae8584caa73bULL;
		h[2] = 0x3c6ef372fe94f82bULL;
		h[3] = 0xa54ff53a5f1d36f1ULL;
		h[4] = 0x510e527fade682d1ULL;
		h[5] = 0x9b05688c2b3e6c1fULL;
		h[6] = 0x1f83d9abfb41bd6bULL;
		h[7] = 0x5be0cd19137e2179ULL;
		*/
#pragma unroll 8
		for(int i=0;i<8;i++)
			h[i] = d_constMem[i];

		// 128 Byte f�r die Message
		uint64_t buf[16];

		// Message f�r die erste Runde in Register holen
#pragma unroll 8
		for (int i=0; i < 8; ++i) buf[i] = inpHash[i];

		/*
		buf[ 8] = 0x0000000000000080ull;
		buf[ 9] = 0;
		buf[10] = 0;
		buf[11] = 0;
		buf[12] = 0;
		buf[13] = 0x0100000000000000ull;
		buf[14] = 0;
		buf[15] = 0x0002000000000000ull;
		*/
#pragma unroll 8
		for(int i=0;i<8;i++)
			buf[i+8] = d_constHashPadding[i];

		// die einzige Hashing-Runde
		quark_blake512_compress( h, buf, c_sigma, c_u512, 512 );

		// Hash rauslassen
#if __CUDA_ARCH__ >= 130
		// ausschliesslich 32 bit Operationen sofern die SM1.3 double intrinsics verf�gbar sind
		uint32_t *outHash = (uint32_t*)&g_hash[8 * hashPosition];
#pragma unroll 8
		for (int i=0; i < 8; ++i) {
			outHash[2*i+0] = cuda_swab32( HIWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( LOWORD(h[i]) );
		}
#else
		// in dieser Version passieren auch ein paar 64 Bit Shifts
		uint64_t *outHash = &g_hash[8 * hashPosition];
#pragma unroll 8
		for (int i=0; i < 8; ++i)
		{
			//outHash[i] = cuda_swab64( h[i] );
			outHash[i] = SWAP64(h[i]);
		}
#endif
	}
}

__global__ void quark_blake512_gpu_hash_80(int threads, uint32_t startNounce, void *outputHash)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Z�hler
		uint32_t nounce = startNounce + thread;

		// State vorbereiten
		uint64_t h[8];
		/*
		h[0] = 0x6a09e667f3bcc908ULL;
		h[1] = 0xbb67ae8584caa73bULL;
		h[2] = 0x3c6ef372fe94f82bULL;
		h[3] = 0xa54ff53a5f1d36f1ULL;
		h[4] = 0x510e527fade682d1ULL;
		h[5] = 0x9b05688c2b3e6c1fULL;
		h[6] = 0x1f83d9abfb41bd6bULL;
		h[7] = 0x5be0cd19137e2179ULL;
		*/
#pragma unroll 8
		for(int i=0;i<8;i++)
			h[i] = d_constMem[i];
		// 128 Byte f�r die Message
		uint64_t buf[16];

		// Message f�r die erste Runde in Register holen
#pragma unroll 16
		for (int i=0; i < 16; ++i) buf[i] = c_PaddedMessage80[i];

		// die Nounce durch die thread-spezifische ersetzen
		buf[9] = REPLACE_HIWORD(buf[9], cuda_swab32(nounce));

		// die einzige Hashing-Runde
		quark_blake512_compress( h, buf, c_sigma, c_u512, 640 );

		// Hash rauslassen
#if __CUDA_ARCH__ >= 130
		// ausschliesslich 32 bit Operationen sofern die SM1.3 double intrinsics verf�gbar sind
		uint32_t *outHash = (uint32_t *)outputHash + 16 * thread;
#pragma unroll 8
		for (int i=0; i < 8; ++i) {
			outHash[2*i+0] = cuda_swab32( HIWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( LOWORD(h[i]) );
		}
#else
		// in dieser Version passieren auch ein paar 64 Bit Shifts
		uint64_t *outHash = (uint64_t *)outputHash + 8 * thread;
#pragma unroll 8
		for (int i=0; i < 8; ++i)
		{
			//outHash[i] = cuda_swab64( h[i] );
			outHash[i] = SWAP64(h[i]);
		}
#endif
	}
}


// ---------------------------- END CUDA quark_blake512 functions ------------------------------------

// Setup-Funktionen
__host__ void quark_blake512_cpu_init(int thr_id, int threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol( HIP_SYMBOL(c_sigma),
						host_sigma,
						sizeof(host_sigma),
						0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol( HIP_SYMBOL(c_u512),
						host_u512,
						sizeof(host_u512),
						0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol( HIP_SYMBOL(d_constMem),
						h_constMem,
						sizeof(h_constMem),
						0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol( HIP_SYMBOL(d_constHashPadding),
						h_constHashPadding,
						sizeof(h_constHashPadding),
						0, hipMemcpyHostToDevice);
}

// Blake512 f�r 80 Byte grosse Eingangsdaten
__host__ void quark_blake512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	PaddedMessage[80] = 0x80;
	PaddedMessage[111] = 1;
	PaddedMessage[126] = 0x02;
	PaddedMessage[127] = 0x80;

	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol( HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__ void quark_blake512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order)
{
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	quark_blake512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_nonceVector, (uint64_t*)d_outputHash);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void quark_blake512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_outputHash, int order)
{
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	quark_blake512_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);
}
