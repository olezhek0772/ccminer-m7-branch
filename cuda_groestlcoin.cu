#include "hip/hip_runtime.h"
// Auf Groestlcoin spezialisierte Version von Groestl inkl. Bitslice

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

// aus cpu-miner.c
extern int device_map[8];

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// Folgende Definitionen sp�ter durch header ersetzen
//////typedef unsigned int uint32_t;
typedef unsigned short uint16_t;
//typedef unsigned int uint32_t;

// diese Struktur wird in der Init Funktion angefordert
static hipDeviceProp_t props[8];

// globaler Speicher f�r alle HeftyHashes aller Threads
__constant__ uint32_t pTarget[8]; // Single GPU
extern uint32_t *d_resultNonce[8];

__constant__ uint32_t groestlcoin_gpu_msg[32];

// 64 Register Variante f�r Compute 3.0
#include "groestl_functions_quad.cu"
#include "bitslice_transformations_quad.cu"

#define SWAB32(x)        ( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )

__global__ void __launch_bounds__(256, 4)
 groestlcoin_gpu_hash_quad(int threads, uint32_t startNounce, uint32_t *resNounce)
{
    // durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    int thread = (blockDim.x * blockIdx.x + threadIdx.x) / 4;
    if (thread < threads)
    {
        // GROESTL
        uint32_t paddedInput[8];
#pragma unroll 8
        for(int k=0;k<8;k++) paddedInput[k] = groestlcoin_gpu_msg[4*k+threadIdx.x%4];

        uint32_t nounce = startNounce + thread;
        if ((threadIdx.x % 4) == 3)
            paddedInput[4] = SWAB32(nounce);  // 4*4+3 = 19

        uint32_t msgBitsliced[8];
        to_bitslice_quad(paddedInput, msgBitsliced);

        uint32_t state[8];
        for (int round=0; round<2; round++)
        {
            groestl512_progressMessage_quad(state, msgBitsliced);

            if (round < 1)
            {
                // Verkettung zweier Runden inclusive Padding.
                msgBitsliced[ 0] = __byte_perm(state[ 0], 0x00800100, 0x4341 + ((threadIdx.x%4)==3)*0x2000);
                msgBitsliced[ 1] = __byte_perm(state[ 1], 0x00800100, 0x4341);
                msgBitsliced[ 2] = __byte_perm(state[ 2], 0x00800100, 0x4341);
                msgBitsliced[ 3] = __byte_perm(state[ 3], 0x00800100, 0x4341);
                msgBitsliced[ 4] = __byte_perm(state[ 4], 0x00800100, 0x4341);
                msgBitsliced[ 5] = __byte_perm(state[ 5], 0x00800100, 0x4341);
                msgBitsliced[ 6] = __byte_perm(state[ 6], 0x00800100, 0x4341);
                msgBitsliced[ 7] = __byte_perm(state[ 7], 0x00800100, 0x4341 + ((threadIdx.x%4)==0)*0x0010);
            }
        }

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t out_state[16];
        from_bitslice_quad(state, out_state);
        
        if (threadIdx.x % 4 == 0)
        {
            int i, position = -1;
            bool rc = true;

    #pragma unroll 8
            for (i = 7; i >= 0; i--) {
                if (out_state[i] > pTarget[i]) {
                    if(position < i) {
                        position = i;
                        rc = false;
                    }
                 }
                 if (out_state[i] < pTarget[i]) {
                    if(position < i) {
                        position = i;
                        rc = true;
                    }
                 }
            }

            if(rc == true)
                if(resNounce[0] > nounce)
                    resNounce[0] = nounce;
        }
    }
}

// Setup-Funktionen
__host__ void groestlcoin_cpu_init(int thr_id, int threads)
{
    hipSetDevice(device_map[thr_id]);

    hipGetDeviceProperties(&props[thr_id], device_map[thr_id]);

    // Speicher f�r Gewinner-Nonce belegen
    hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)); 
}

__host__ void groestlcoin_cpu_setBlock(int thr_id, void *data, void *pTargetIn)
{
    // Nachricht expandieren und setzen
    uint32_t msgBlock[32];

    memset(msgBlock, 0, sizeof(uint32_t) * 32);
    memcpy(&msgBlock[0], data, 80);

    // Erweitere die Nachricht auf den Nachrichtenblock (padding)
    // Unsere Nachricht hat 80 Byte
    msgBlock[20] = 0x80;
    msgBlock[31] = 0x01000000;

    // groestl512 braucht hierf�r keinen CPU-Code (die einzige Runde wird
    // auf der GPU ausgef�hrt)

    // Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
    hipMemcpyToSymbol(HIP_SYMBOL( groestlcoin_gpu_msg),
                        msgBlock,
                        128);

    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL( pTarget),
                        pTargetIn,
                        sizeof(uint32_t) * 8 );
}

__host__ void groestlcoin_cpu_hash(int thr_id, int threads, uint32_t startNounce, void *outputHashes, uint32_t *nounce)
{
    int threadsperblock = 256;

    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    int factor = 4;

        // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
    groestlcoin_gpu_hash_quad<<<grid, block, shared_size>>>(threads, startNounce, d_resultNonce[thr_id]);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, 0, thr_id);

    hipMemcpy(nounce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}
